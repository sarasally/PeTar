#include "hip/hip_runtime.h"
//#include "class.hpp"
//#include "force.hpp"
#include<particle_simulator.hpp>
#include "cuda_pointer.h"
#include "force_gpu_cuda.hpp"

#ifdef GPU_PROFILE
GPUProfile gpu_profile;
GPUCounter gpu_counter;
#endif

enum{
	N_THREAD_GPU = 32,
	N_WALK_LIMIT = 1000,
	NI_LIMIT     = N_WALK_LIMIT*1000,
	NJ_LIMIT     = N_WALK_LIMIT*10000,
};

struct EpiGPU{
	float3 pos;
    float  r_search;
	int    id_walk;
};

struct EpiDev{
	float3 pos;
    float  r_search;
};

struct EpjGPU{
	float3 pos;
    float  m;
    float  r_search;
};

struct SpjGPU{
    float3 pos;
    float  m;
#ifdef USE_QUAD
    float  qxx, qyy, qzz, qxy, qxz, qyz;
#endif
};

struct ForceGPU{
	float4 accp;
    int    nnb;
};

//! device pair force of Epi and Epi with linear cutoff
inline __device__ ForceGPU dev_gravity_ep_ep(
    float  eps2,
    float  rcut2,
    float  G,
    EpiDev epii,
    EpjGPU epjj,
    ForceGPU forcei)
{
	float dx = epjj.pos.x - epii.pos.x;
	float dy = epjj.pos.y - epii.pos.y;
	float dz = epjj.pos.z - epii.pos.z;

	float r2   = eps2 + dx*dx + dy*dy + dz*dz;
    float rsmin = max(epii.r_search, epjj.r_search);
    if (r2 < rsmin*rsmin) forcei.nnb ++;

    float r2_cut = (r2 > rcut2)? r2 : rcut2;
	float rinv = rsqrtf(r2_cut);
	float pij  = epjj.m * rinv;
	float mri3 = G*rinv*rinv * pij;

	forcei.accp.x += mri3 * dx;
	forcei.accp.y += mri3 * dy;
	forcei.accp.z += mri3 * dz;
	forcei.accp.w -= G*pij;

    return forcei;
}

__device__ ForceGPU force_kernel_ep_ep_1walk(
    EpjGPU       *jpsh,
    const EpiDev  epii,
    const int     id_walk,
    const int3   *ij_disp,
    const EpjGPU *epj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_epj,
#endif
    ForceGPU      forcei,
    const float   eps2,
    const float   rcut2,
    const float   G) {

    const int tid = threadIdx.x;
    const int j_head = ij_disp[id_walk  ].y;
    const int j_tail = ij_disp[id_walk+1].y;

	for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
		// __syncthreads();
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		jpsh[tid] = epj[id_epj[j+tid]];
#else
		jpsh[tid] = ((EpjGPU *)(epj + j)) [tid];
#endif
		// __syncthreads();

		if(j_tail-j < N_THREAD_GPU){
			for(int jj=0; jj<j_tail-j; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[jj], forcei);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[jj], forcei);
			}
		}
	}
	
	return forcei;
}

__device__ ForceGPU force_kernel_ep_ep_2walk(
    EpjGPU        jpsh[2][N_THREAD_GPU],
    const EpiDev  epii,
    const int     id_walk,
    const int     iwalk0,
    const int     iwalk1,
    const int3   *ij_disp,
    const EpjGPU *epj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_epj,
#endif
    ForceGPU      forcei,
    const float   eps2,
    const float   rcut2,
    const float   G) {

	const int jbeg0 = ij_disp[iwalk0].y;
	const int jbeg1 = ij_disp[iwalk1].y;
	const int jend0 = ij_disp[iwalk0 + 1].y;
	const int jend1 = ij_disp[iwalk1 + 1].y;
	const int nj0   = jend0 - jbeg0;
	const int nj1   = jend1 - jbeg1;

	const int nj_longer  = nj0 > nj1 ? nj0 : nj1;
	const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
	const int walk_longer= nj0 > nj1 ? 0 : 1;
	const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

	const int mywalk = id_walk==iwalk0 ? 0 : 1;

    const int tid = threadIdx.x;
	for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		jpsh[0][tid] = epj[id_epj[jbeg0 + j + tid]];
		jpsh[1][tid] = epj[id_epj[jbeg1 + j + tid]];
#else
		jpsh[0][tid] = ((EpjGPU *)(epj + jbeg0 + j)) [tid];
		jpsh[1][tid] = ((EpjGPU *)(epj + jbeg1 + j)) [tid];
#endif
		if(nj_shorter-j < N_THREAD_GPU){
			for(int jj=0; jj<nj_shorter-j; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[mywalk][jj], forcei);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[mywalk][jj], forcei);
			}
		}
	}
	for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		jpsh[0][tid] = epj[id_epj[jbeg_longer + j + tid]];
#else
		jpsh[0][tid] = ((EpjGPU *)(epj + jbeg_longer +  j)) [tid];
#endif
		int jrem = nj_longer - j;
		if(jrem < N_THREAD_GPU){
			for(int jj=0; jj<jrem; jj++){
				if(mywalk == walk_longer)
                    forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[0][jj], forcei);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				if(mywalk == walk_longer)
                    forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[0][jj], forcei);
			}
		}
	}

	return forcei;
}

__device__ ForceGPU force_kernel_ep_ep_multiwalk(
    const EpiDev  epii,
    const int     id_walk,
    const int3   *ij_disp,
    const EpjGPU *epj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_epj,
#endif
    ForceGPU      forcei,
    const float   eps2,
    const float   rcut2,
    const float   G) {

    const int j_head = ij_disp[id_walk  ].y;
    const int j_tail = ij_disp[id_walk+1].y;

    for(int j=j_head; j<j_tail; j++){
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		EpjGPU epjj = epj[id_epj[j]];
#else
		EpjGPU epjj = epj[j];
#endif
		forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, epjj, forcei);
	}
	return forcei;
}

__global__ void force_kernel_ep_ep(
    const int3   * ij_disp,
    const EpiGPU * epi,
    const EpjGPU * epj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_epj,
#endif
    ForceGPU     * force,
    const float    eps2,
    const float    rcut2,
    const float    G) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	EpiDev epii;
    epii.pos       = epi[tid].pos;
    epii.r_search  = epi[tid].r_search;
	int    id_walk = epi[tid].id_walk;
	ForceGPU forcei;
    forcei.accp = make_float4(0.f, 0.f, 0.f, 0.f);
    forcei.nnb  = 0;

	int t_head = blockDim.x * blockIdx.x;
	int t_tail = t_head + N_THREAD_GPU - 1;
	int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);

	__shared__ EpjGPU jpsh[2][N_THREAD_GPU];

#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
	if(1 == nwalk_in_block){
		forcei = force_kernel_ep_ep_1walk(jpsh[0], epii, id_walk, ij_disp, epj, id_epj, forcei, eps2, rcut2, G);
	} else if(2 == nwalk_in_block){
		// accp = force_kernel_ep_ep_multiwalk(epii, id_walk, ij_disp, epj, accp, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		forcei = force_kernel_ep_ep_2walk(jpsh, epii, id_walk, iwalk0, iwalk1, ij_disp, epj, id_epj, forcei, eps2, rcut2, G);
	} else{
		forcei = force_kernel_ep_ep_multiwalk(epii, id_walk, ij_disp, epj, id_epj, forcei, eps2, rcut2, G);
	}
#else
	if(1 == nwalk_in_block){
		forcei = force_kernel_ep_ep_1walk(jpsh[0], epii, id_walk, ij_disp, epj, forcei, eps2, rcut2, G);
	} else if(2 == nwalk_in_block){
		// accp = force_kernel_ep_ep_multiwalk(epii, id_walk, ij_disp, epj, accp, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		forcei = force_kernel_ep_ep_2walk(jpsh, epii, id_walk, iwalk0, iwalk1, ij_disp, epj, forcei, eps2, rcut2, G);
	} else{
		forcei = force_kernel_ep_ep_multiwalk(epii, id_walk, ij_disp, epj, forcei, eps2, rcut2, G);
	}
#endif
	force[tid] = forcei;
}

//! device pair force of Epi and Spi 
inline __device__ float4 dev_gravity_ep_sp(
    float  eps2,
    float  G,
    float3 posi,
    SpjGPU spjj,
    float4 accpi) {

	float dx = posi.x - spjj.pos.x;
	float dy = posi.y - spjj.pos.y;
	float dz = posi.z - spjj.pos.z;

	float r2   = eps2 + dx*dx + dy*dy + dz*dz;
	float rinv = rsqrtf(r2);

#ifdef USE_QUAD
    float qrx = spjj.qxx*dx + spjj.qxy*dy + spjj.qxz*dz;
    float qry = spjj.qxy*dx + spjj.qyy*dy + spjj.qyz*dz;
    float qrz = spjj.qxz*dx + spjj.qyz*dy + spjj.qzz*dz;
    float tr = spjj.qxx + spjj.qyy + spjj.qzz;
    
    float qrr = qrx*dx + qry*dy + qrz*dz;
    float rinv2 = rinv*rinv;
    float rinv3 = rinv2*rinv;
    float rinv5 = rinv2*rinv3*1.5f;
    float qrr_r5 = rinv5*qrr;
    float qrr_r7 = rinv2*qrr_r5;
    float A = G*(spjj.m*rinv3 - tr*rinv5 + 5.0f*qrr_r7);
    float B = -2.0f*G*rinv5;
    
    accpi.x -= A*dx + B*qrx;
    accpi.y -= A*dy + B*qry;
    accpi.z -= A*dz + B*qrz;
    accpi.w -= G*(spjj.m*rinv - 0.5f*tr*rinv3 + qrr_r5);

#else
    
	float pij  = spjj.m * rinv;
	float mri3 = G*rinv*rinv * pij;

	accpi.x += mri3 * dx;
	accpi.y += mri3 * dy;
	accpi.z += mri3 * dz;
	accpi.w -= G*pij;

#endif

    return accpi;
}

__device__ float4 force_kernel_ep_sp_1walk(
    SpjGPU   *jpsh,
    const float3  posi,
    const int     id_walk,
    const int3   *ij_disp,
    const SpjGPU *spj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_spj,
#endif
    float4        accpi,
    const float   eps2,
    const float   G) {

    const int tid = threadIdx.x;
    const int j_head = ij_disp[id_walk  ].z;
    const int j_tail = ij_disp[id_walk+1].z;

	for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
		// __syncthreads();
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		jpsh[tid] = spj[id_spj[j + tid]];
#else
		jpsh[tid] = ((SpjGPU *)(spj + j)) [tid];
#endif
		// __syncthreads();

		if(j_tail-j < N_THREAD_GPU){
			for(int jj=0; jj<j_tail-j; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[jj], accpi);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[jj], accpi);
			}
		}
	}
	
	return accpi;
}

__device__ float4 force_kernel_ep_sp_2walk(
    SpjGPU        jpsh[2][N_THREAD_GPU],
    const float3  posi,
    const int     id_walk,
    const int     iwalk0,
    const int     iwalk1,
    const int3   *ij_disp,
    const SpjGPU *spj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_spj,
#endif
    float4        accpi,
    const float   eps2,
    const float   G)
{
	const int jbeg0 = ij_disp[iwalk0].z;
	const int jbeg1 = ij_disp[iwalk1].z;
	const int jend0 = ij_disp[iwalk0 + 1].z;
	const int jend1 = ij_disp[iwalk1 + 1].z;
	const int nj0   = jend0 - jbeg0;
	const int nj1   = jend1 - jbeg1;

	const int nj_longer  = nj0 > nj1 ? nj0 : nj1;
	const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
	const int walk_longer= nj0 > nj1 ? 0 : 1;
	const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

	const int mywalk = id_walk==iwalk0 ? 0 : 1;

    const int tid = threadIdx.x;
	for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		jpsh[0][tid] = spj[id_spj[jbeg0 + j + tid]];
		jpsh[1][tid] = spj[id_spj[jbeg1 + j + tid]];
#else
		jpsh[0][tid] = ((SpjGPU *)(spj + jbeg0 + j)) [tid];
		jpsh[1][tid] = ((SpjGPU *)(spj + jbeg1 + j)) [tid];
#endif
		if(nj_shorter-j < N_THREAD_GPU){
			for(int jj=0; jj<nj_shorter-j; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[mywalk][jj], accpi);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[mywalk][jj], accpi);
			}
		}
	}
	for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		jpsh[0][tid] = spj[id_spj[jbeg_longer + j + tid]];
#else
		jpsh[0][tid] = ((SpjGPU *)(spj + jbeg_longer +  j)) [tid];
#endif
		int jrem = nj_longer - j;
		if(jrem < N_THREAD_GPU){
			for(int jj=0; jj<jrem; jj++){
				if(mywalk == walk_longer)
                    accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[0][jj], accpi);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				if(mywalk == walk_longer)
                    accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[0][jj], accpi);
			}
		}
	}

	return accpi;
}

__device__ float4 force_kernel_ep_sp_multiwalk(
    const float3  posi,
    const int     id_walk,
    const int3   *ij_disp,
    const SpjGPU *spj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_spj,
#endif
    float4        accpi,
    const float   eps2,
    const float   G)
{
    const int j_head = ij_disp[id_walk  ].z;
    const int j_tail = ij_disp[id_walk+1].z;

    for(int j=j_head; j<j_tail; j++){
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
		SpjGPU spjj = spj[id_spj[j]];
#else        
		SpjGPU spjj = spj[j];
#endif
		accpi = dev_gravity_ep_sp(eps2, G, posi, spjj, accpi);
	}
	return accpi;
}

__global__ void force_kernel_ep_sp(
    const int3   * ij_disp,
    const EpiGPU * epi,
    const SpjGPU * spj, 
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    const int    *id_spj,
#endif
    ForceGPU     * force,
    const float    eps2,
    const float    G) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float3 posi    = epi[tid].pos;
	int    id_walk = epi[tid].id_walk;
	float4 accpi   = force[tid].accp;

	int t_head = blockDim.x * blockIdx.x;
	int t_tail = t_head + N_THREAD_GPU - 1;
	int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);

	__shared__ SpjGPU jpsh[2][N_THREAD_GPU];

#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
	if(1 == nwalk_in_block){
		accpi = force_kernel_ep_sp_1walk(jpsh[0], posi, id_walk, ij_disp, spj, id_spj, accpi, eps2, G);
	} else if(2 == nwalk_in_block){
		// accpi = force_kernel_ep_sp_multiwalk(posi, id_walk, ij_disp, spj, accpi, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		accpi = force_kernel_ep_sp_2walk(jpsh, posi, id_walk, iwalk0, iwalk1, ij_disp, spj, id_spj, accpi, eps2, G);
	} else{
		accpi = force_kernel_ep_sp_multiwalk(posi, id_walk, ij_disp, spj, id_spj, accpi, eps2, G);
	}
#else
	if(1 == nwalk_in_block){
		accpi = force_kernel_ep_sp_1walk(jpsh[0], posi, id_walk, ij_disp, spj, accpi, eps2, G);
	} else if(2 == nwalk_in_block){
		// accpi = force_kernel_ep_sp_multiwalk(posi, id_walk, ij_disp, spj, accpi, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		accpi = force_kernel_ep_sp_2walk(jpsh, posi, id_walk, iwalk0, iwalk1, ij_disp, spj, accpi, eps2, G);
	} else{
		accpi = force_kernel_ep_sp_multiwalk(posi, id_walk, ij_disp, spj, accpi, eps2, G);
    }
#endif
	force[tid].accp = accpi;
}

static cudaPointer<EpiGPU>    dev_epi;
static cudaPointer<EpjGPU>    dev_epj;
static cudaPointer<SpjGPU>    dev_spj;
static cudaPointer<ForceGPU>  dev_force;
static cudaPointer<int3>      ij_disp;

static bool init_call = true;
#ifdef GPU_PROFILE
static hipEvent_t cu_event_sends;
static hipEvent_t cu_event_sendf;
static hipEvent_t cu_event_disp;
static hipEvent_t cu_event_htod;
static hipEvent_t cu_event_calc;
static hipEvent_t cu_event_retr;
static hipEvent_t cu_event_dtoh;
#endif

#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX

static cudaPointer<int>      dev_id_epj;
static cudaPointer<int>      dev_id_spj;

PS::S32 CalcForceWithLinearCutoffCUDAMultiWalk::operator()(const PS::S32 tag,
                                                           const PS::S32 n_walk,
                                                           const EPISoft ** epi,
                                                           const PS::S32 *  n_epi,
                                                           const PS::S32 ** id_epj,
                                                           const PS::S32 *  n_epj,
                                                           const PS::S32 ** id_spj,
                                                           const PS::S32 *  n_spj,
                                                           const EPJSoft * epj,
                                                           const PS::S32 n_epj_tot,
                                                           const SPJSoft * spj,
                                                           const PS::S32 n_spj_tot,
                                                           const bool send_flag) {
    assert(n_walk <= N_WALK_LIMIT);
    if(init_call){
        int ngpu;
        hipGetDeviceCount(&ngpu);
        int device_index=my_rank % ngpu;
        hipSetDevice(device_index);
        //std::cerr<<"MPI rank "<<my_rank<<" set GPU device "<<device_index<<std::endl;
        
        dev_epi  .allocate(NI_LIMIT);
        dev_epj  .allocate(NJ_LIMIT);
        dev_spj  .allocate(NJ_LIMIT);
        dev_force.allocate(NI_LIMIT);
        ij_disp  .allocate(N_WALK_LIMIT+2);
        dev_id_epj .allocate(NJ_LIMIT);
        dev_id_spj .allocate(NJ_LIMIT);
        hipEventCreate(&cu_event_sends);
        hipEventCreate(&cu_event_sendf);
        hipEventCreate(&cu_event_disp);
        hipEventCreate(&cu_event_htod);
        hipEventCreate(&cu_event_calc);
        hipEventCreate(&cu_event_retr);
        hipEventCreate(&cu_event_dtoh);
        init_call = false;
    }

    //const float eps2 = EPISoft::eps * EPISoft::eps;
    //const PS::F64 rcut2 = EPISoft::r_out*EPISoft::r_out;
    //const PS::F64 G = ForceSoft::grav_const;

    if(send_flag==true){
#ifdef GPU_PROFILE
        gpu_profile.copy.start();
#endif
        /*
          if(dev_epj.size < n_epj_tot+n_spj_tot){
          dev_epj.free();
          dev_epj.allocate(n_epj_tot+n_spj_tot);
          }
        */
#pragma omp parallel for
        for(PS::S32 i=0; i<n_epj_tot; i++){
            dev_epj[i].pos.x  = epj[i].pos.x;
            dev_epj[i].pos.y  = epj[i].pos.y;
            dev_epj[i].pos.z  = epj[i].pos.z;
            dev_epj[i].m      = epj[i].mass;
            dev_epj[i].r_search = epj[i].r_search;
        }
#pragma omp parallel for
        for(PS::S32 i=0; i<n_spj_tot; i++){
            dev_spj[i].pos.x  = spj[i].pos.x;
            dev_spj[i].pos.y  = spj[i].pos.y;
            dev_spj[i].pos.z  = spj[i].pos.z;
            dev_spj[i].m      = spj[i].mass;
#ifdef USE_QUAD
            dev_spj[i].qxx    = spj[i].quad.xx;
            dev_spj[i].qyy    = spj[i].quad.yy;
            dev_spj[i].qzz    = spj[i].quad.zz;
            dev_spj[i].qxy    = spj[i].quad.xy;
            dev_spj[i].qxz    = spj[i].quad.xz;
            dev_spj[i].qyz    = spj[i].quad.yz;
#endif
        }
#ifdef GPU_PROFILE
        gpu_profile.copy.end();
        hipEventRecord(cu_event_sends);
#endif
        dev_epj.htod(n_epj_tot);
        dev_spj.htod(n_spj_tot);
#ifdef GPU_PROFILE
        hipEventRecord(cu_event_sendf);
#endif
        return 0;
    }
    else{
#ifdef GPU_PROFILE
        gpu_profile.copy.start();
#endif
        ij_disp[0].x = 0;
        ij_disp[0].y = 0;
        ij_disp[0].z = 0;
        for(int k=0; k<n_walk; k++){
            ij_disp[k+1].x = ij_disp[k].x + n_epi[k];
            ij_disp[k+1].y = ij_disp[k].y + n_epj[k];
            ij_disp[k+1].z = ij_disp[k].z + n_spj[k];
        }
        ij_disp[n_walk+1] = ij_disp[n_walk];
        assert(ij_disp[n_walk].x < NI_LIMIT);
        assert(ij_disp[n_walk].y < NJ_LIMIT);
        assert(ij_disp[n_walk].z < NJ_LIMIT);

        int ni_tot_reg = ij_disp[n_walk].x;
        if(ni_tot_reg % N_THREAD_GPU){
            ni_tot_reg /= N_THREAD_GPU;
            ni_tot_reg++;
            ni_tot_reg *= N_THREAD_GPU;
        }

        int ni_tot = ij_disp[n_walk].x;
        int nej_tot = ij_disp[n_walk].y;
        int nsj_tot = ij_disp[n_walk].z;

#pragma omp parallel for schedule(dynamic)
        for(int iw=0; iw<n_walk; iw++){
            for(int i=0; i<n_epi[iw]; i++){
                int ik = i+ij_disp[iw].x;
                dev_epi[ik].pos.x = epi[iw][i].pos.x;
                dev_epi[ik].pos.y = epi[iw][i].pos.y;
                dev_epi[ik].pos.z = epi[iw][i].pos.z;
                dev_epi[ik].r_search = epi[iw][i].r_search;
                dev_epi[ik].id_walk = iw;
            }
            for(int j=0; j<n_epj[iw]; j++){
                int jk = j+ij_disp[iw].y;
                dev_id_epj[jk] = id_epj[iw][j];
            }
            for(int j=0; j<n_spj[iw]; j++){
                int jk = j+ij_disp[iw].z;
                dev_id_spj[jk] = id_spj[iw][j];
            }
        }
        for(int i=ni_tot; i<ni_tot_reg; i++){
            dev_epi[i].id_walk = n_walk;
        }

#ifdef GPU_PROFILE
        gpu_profile.copy.end();
        gpu_counter.n_walk+= n_walk;
        gpu_counter.n_epi += ni_tot;
        gpu_counter.n_epj += nej_tot;
        gpu_counter.n_spj += nsj_tot;
        gpu_counter.n_call+= 1;
        hipEventRecord(cu_event_disp);
#endif
        ij_disp.htod(n_walk + 2);
        dev_epi.htod(ni_tot_reg);
        dev_id_epj.htod(nej_tot);
        dev_id_spj.htod(nsj_tot);

#ifdef GPU_PROFILE
        hipEventRecord(cu_event_htod);
#endif

        int nblocks  = ni_tot_reg / N_THREAD_GPU;
        int nthreads = N_THREAD_GPU;
        force_kernel_ep_ep <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_id_epj, dev_force, eps2, rcut2, G);
        force_kernel_ep_sp <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_spj, dev_id_spj, dev_force, eps2, G);

#ifdef GPU_PROFILE
        hipEventRecord(cu_event_calc);
#endif
        return 0;
    }
}

#else

PS::S32 CalcForceWithLinearCutoffCUDA::operator()(const PS::S32  tag,
                                                  const PS::S32  n_walk,
                                                  const EPISoft *epi[],
                                                  const PS::S32  n_epi[],
                                                  const EPJSoft  *epj[],
                                                  const PS::S32  n_epj[],
                                                  const SPJSoft *spj[],
                                                  const PS::S32  n_spj[]){
    assert(n_walk <= N_WALK_LIMIT);
    if(init_call){
        int ngpu;
        hipGetDeviceCount(&ngpu);
        int device_index=my_rank % ngpu;
        hipSetDevice(device_index);
        //std::cerr<<"MPI rank "<<my_rank<<"set GPU device "<<device_index<<std::endl;
        
        dev_epi  .allocate(NI_LIMIT);
        dev_epj  .allocate(NJ_LIMIT);
        dev_spj  .allocate(NJ_LIMIT);
        dev_force.allocate(NI_LIMIT);
        ij_disp  .allocate(N_WALK_LIMIT+2);
        hipEventCreate(&cu_event_disp);
        hipEventCreate(&cu_event_htod);
        hipEventCreate(&cu_event_calc);
        hipEventCreate(&cu_event_retr);
        hipEventCreate(&cu_event_dtoh);
        init_call = false;
    }
#ifdef GPU_PROFILE
    gpu_profile.copy.start();
#endif
    //const float eps2 = EPISoft::eps * EPISoft::eps;
    //const PS::F64 rcut2 = EPISoft::r_out*EPISoft::r_out;
    //const PS::F64 G = ForceSoft::grav_const;
    ij_disp[0].x = 0;
    ij_disp[0].y = 0;
    ij_disp[0].z = 0;
    for(int k=0; k<n_walk; k++){
        ij_disp[k+1].x = ij_disp[k].x + n_epi[k];
        ij_disp[k+1].y = ij_disp[k].y + n_epj[k];
        ij_disp[k+1].z = ij_disp[k].z + n_spj[k];
    }
    ij_disp[n_walk+1] = ij_disp[n_walk];

    assert(ij_disp[n_walk].x < NI_LIMIT);
    assert(ij_disp[n_walk].y < NJ_LIMIT);
    assert(ij_disp[n_walk].z < NJ_LIMIT);

    int ni_tot_reg = ij_disp[n_walk].x;
    if(ni_tot_reg % N_THREAD_GPU){
        ni_tot_reg /= N_THREAD_GPU;
        ni_tot_reg++;
        ni_tot_reg *= N_THREAD_GPU;
    }

    int ni_tot = ij_disp[n_walk].x;
    int nej_tot = ij_disp[n_walk].y;
    int nsj_tot = ij_disp[n_walk].z;

#pragma omp parallel for schedule(dynamic)
    for(int iw=0; iw<n_walk; iw++){
        for(int i=0; i<n_epi[iw]; i++){
            int ik = i+ij_disp[iw].x;
            dev_epi[ik].pos.x = epi[iw][i].pos.x;
            dev_epi[ik].pos.y = epi[iw][i].pos.y;
            dev_epi[ik].pos.z = epi[iw][i].pos.z;
            dev_epi[ik].r_search = epi[iw][i].r_search;
            dev_epi[ik].id_walk = iw;
        }
        for(int j=0; j<n_epj[iw]; j++){
            int jk = j+ij_disp[iw].y;
            dev_epj[jk].pos.x  = epj[iw][j].pos.x;
            dev_epj[jk].pos.y  = epj[iw][j].pos.y;
            dev_epj[jk].pos.z  = epj[iw][j].pos.z;
            dev_epj[jk].m      = epj[iw][j].mass;
            dev_epj[jk].r_search = epj[iw][j].r_search;
        }
        for(int j=0; j<n_spj[iw]; j++){
            int jk = j+ij_disp[iw].z;
            dev_spj[jk].pos.x  = spj[iw][j].pos.x;
            dev_spj[jk].pos.y  = spj[iw][j].pos.y;
            dev_spj[jk].pos.z  = spj[iw][j].pos.z;
            dev_spj[jk].m      = spj[iw][j].getCharge();
#ifdef USE_QUAD
            dev_spj[jk].qxx    = spj[iw][j].quad.xx;
            dev_spj[jk].qyy    = spj[iw][j].quad.yy;
            dev_spj[jk].qzz    = spj[iw][j].quad.zz;
            dev_spj[jk].qxy    = spj[iw][j].quad.xy;
            dev_spj[jk].qxz    = spj[iw][j].quad.xz;
            dev_spj[jk].qyz    = spj[iw][j].quad.yz;
#endif
        }
    }
    for(int i=ni_tot; i<ni_tot_reg; i++){
        dev_epi[i].id_walk = n_walk;
    }

#ifdef GPU_PROFILE
    gpu_profile.copy.end();
    gpu_counter.n_walk+= n_walk;
    gpu_counter.n_epi += ni_tot;
    gpu_counter.n_epj += nej_tot;
    gpu_counter.n_spj += nsj_tot;
    gpu_counter.n_call+= 1;
    hipEventRecord(cu_event_disp);
#endif
    ij_disp.htod(n_walk + 2);
    dev_epi.htod(ni_tot_reg);
    dev_epj.htod(nej_tot);
    dev_spj.htod(nsj_tot);

#ifdef GPU_PROFILE
    hipEventRecord(cu_event_htod);
#endif
    int nblocks  = ni_tot_reg / N_THREAD_GPU;
    int nthreads = N_THREAD_GPU;
    force_kernel_ep_ep <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_force, eps2, rcut2, G);
    force_kernel_ep_sp <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_spj, dev_force, eps2, G);

#ifdef GPU_PROFILE
    hipEventRecord(cu_event_calc);
#endif
    return 0;
}

#endif

PS::S32 RetrieveForceCUDA(const PS::S32 tag,
                          const PS::S32 n_walk,
                          const PS::S32 ni[],
                          ForceSoft    *force[]) {

#ifdef GPU_PROFILE
    hipEventRecord(cu_event_retr);
#endif
    int ni_tot = 0;
    for(int k=0; k<n_walk; k++){
        ni_tot += ni[k];
    }
    dev_force.dtoh(ni_tot);

#ifdef GPU_PROFILE
    hipEventRecord(cu_event_dtoh);
    hipEventSynchronize(cu_event_dtoh);
    float send_time=0, calc_time=0, recv_time=0;
    hipEventElapsedTime(&send_time, cu_event_disp, cu_event_htod);
    hipEventElapsedTime(&calc_time, cu_event_htod, cu_event_calc);
    hipEventElapsedTime(&recv_time, cu_event_retr, cu_event_dtoh);
    gpu_profile.send.time += 0.001f*send_time;
    gpu_profile.calc.time += 0.001f*calc_time;
    gpu_profile.recv.time += 0.001f*recv_time;
#ifdef PARTICLE_SIMULATOR_GPU_MULIT_WALK_INDEX
    send_time=0;
    hipEventElapsedTime(&send_time, cu_event_sends, cu_event_sendf);
    hipEventRecord(cu_event_sends);
    hipEventRecord(cu_event_sendf);
    gpu_profile.send.time += 0.001f*send_time;
#endif
    gpu_profile.copy.start();
#endif

    int n_cnt = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int i=0; i<ni[iw]; i++){
            force[iw][i].acc.x = dev_force[n_cnt].accp.x;
            force[iw][i].acc.y = dev_force[n_cnt].accp.y;
            force[iw][i].acc.z = dev_force[n_cnt].accp.z;
            force[iw][i].pot   = dev_force[n_cnt].accp.w;
            force[iw][i].n_ngb = dev_force[n_cnt].nnb;
            n_cnt++;
        }
    }
#ifdef GPU_PROFILE
    gpu_profile.copy.end();
#endif
    return 0;
}
